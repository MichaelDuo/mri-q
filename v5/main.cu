#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

/*
 * C code for creating the Q data structure for fast convolution-based
 * Hessian multiplication for arbitrary k-space trajectories.
 *
 * Inputs:
 * kx - VECTOR of kx values, same length as ky and kz
 * ky - VECTOR of ky values, same length as kx and kz
 * kz - VECTOR of kz values, same length as kx and ky
 * x  - VECTOR of x values, same length as y and z
 * y  - VECTOR of y values, same length as x and z
 * z  - VECTOR of z values, same length as x and y
 * phi - VECTOR of the Fourier transform of the spatial basis
 *      function, evaluated at [kx, ky, kz].  Same length as kx, ky, and kz.
 *
 * recommended g++ options:
 *  -O3 -lm -ffast-math -funroll-all-loops
 */

 #include <stdio.h>
 #include <math.h>
 #include <stdlib.h>
 #include <string.h>
 #include <sys/time.h>
 #include <malloc.h>
 
 #include "parboil.h"
 
 #include "file.h"
 #include "computeQ.cu"
 
 #define FATAL(msg, ...) \
     do {\
         fprintf(stderr, "[%s:%d] "msg"\n", __FILE__, __LINE__, ##__VA_ARGS__);\
         exit(-1);\
     } while(0)
 
 
 int main (int argc, char *argv[])
 {
   int numX, numK;		/* Number of X and K values */
   int original_numK;		/* Number of K values in input file */
   float *kx, *ky, *kz;		/* K trajectory (3D vectors) */
   float *x, *y, *z;		/* X coordinates (3D vectors) */
   float *phiR, *phiI;		/* Phi values (complex) */
   float *phiMag;		/* Magnitude of Phi */
   float *Qr, *Qi;		/* Q signal (complex) */
   struct kValues* kVals;
 
   float *phiR_d, *phiI_d, *phiMag_d;
   float *Qr_d, *Qi_d;
   float *x_d, *y_d, *z_d;
 
   struct pb_Parameters *params;
   struct pb_TimerSet timers;
 
   pb_InitializeTimerSet(&timers);
 
   /* Read command line */
   params = pb_ReadParameters(&argc, argv);
   if ((params->inpFiles[0] == NULL) || (params->inpFiles[1] != NULL))
   {
 
     fprintf(stderr, "Expecting one input filename\n");
     exit(-1);
   }
 
   /* Read in data */
   pb_SwitchToTimer(&timers, pb_TimerID_IO);
   inputData(params->inpFiles[0],
       &original_numK, &numX,
       &kx, &ky, &kz,
       &x, &y, &z,
       &phiR, &phiI);
 
   /* Reduce the number of k-space samples if a number is given
    * on the command line */
   if (argc < 2)
     numK = original_numK;
   else
   {
     int inputK;
     char *end;
     inputK = strtol(argv[1], &end, 10);
     if (end == argv[1])
     {
       fprintf(stderr, "Expecting an integer parameter\n");
       exit(-1);
     }
 
     numK = MIN(inputK, original_numK);
   }
 
   printf("%d pixels in output; %d samples in trajectory; using %d samples\n",
          numX, original_numK, numK);
 
   pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
 
   createDataStructsCPU(numK, numX, &phiMag, &Qr, &Qi);
  
   hipSetDevice(1);
 
   pb_SwitchToTimer(&timers, pb_TimerID_COPY);
   // Allocating memory
   hipMalloc((void** )&phiR_d, sizeof(float) * numK);
   hipMalloc((void** )&phiI_d, sizeof(float) * numK);
   hipMalloc((void** )&phiMag_d, sizeof(float) * numK);
   hipDeviceSynchronize();
 
     /* Allocate pinned memory */
     float *phiR_p, *phiI_p, *phiMag_p;
 
     hipHostAlloc((void **)&phiR_p, sizeof(float) * numK, hipHostMallocDefault);
     hipHostAlloc((void **)&phiI_p, sizeof(float) * numK, hipHostMallocDefault);
     hipHostAlloc((void **)&phiMag_p, sizeof(float) * numK, hipHostMallocDefault);
     for (int i = 0; i < numK; i++) {
       phiR_p[i] = phiR[i];
       phiI_p[i] = phiI[i];
     }
 
 
     int n = 0, i = 0;
     unsigned int offset = 0;
     /* choosing an appropriate number of streams */
     /* least number of samples in the given dataset
        is 2048, BLOCK_SIZE chosen in 512. 2048/4 = 512
        so that each stream has sufficient data to work on */
     const unsigned int n_streams_phimag = 4;
 
     /* divide input data into segments based on number of streams*/
     const unsigned int stream_size = ceil(numK / n_streams_phimag);
 
     hipStream_t stream[n_streams_phimag];
 
     /* Create CUDA streams */
     for (i = 0; i < n_streams_phimag; i++) {
       hipStreamCreate(&stream[i]);
     }
 
     pb_SwitchToTimer(&timers, pb_TimerID_COPY_ASYNC);
     /* Copy data to the device asynchronously */
     for (n = 0; n < n_streams_phimag; n++) {
       offset = n * stream_size;
 
       hipMemcpyAsync(&phiR_d[offset], &phiR_p[offset],
         sizeof(float) * stream_size, hipMemcpyHostToDevice);
       hipMemcpyAsync(&phiI_d[offset], &phiI_p[offset],
         sizeof(float) * stream_size, hipMemcpyHostToDevice);
       hipMemsetAsync(&phiMag_d[offset], 0,
         sizeof(float) * stream_size, stream[n]);
     }
 
     pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);
     /* Call the kernels with respective streams */
     for (n = 0; n < n_streams_phimag; n++) {
       offset = n * stream_size;
       ComputePhiMagGPUAsync(numK, stream_size, phiR_d, phiI_d, phiMag_d,
         stream[n], offset);
     }
 
     pb_SwitchToTimer(&timers, pb_TimerID_COPY_ASYNC);
     /* Copy data from the device asynchronously */
     for (n = 0; n < n_streams_phimag; n++) {
       offset = n * stream_size;
 
       hipMemcpyAsync(&phiMag_p[offset], &phiMag_d[offset],
         sizeof(float) * stream_size, hipMemcpyDeviceToHost);
     }
 
     /* Wait for all streams to finish */
     hipDeviceSynchronize();
 
     /* Delete the streams */
     for (i = 0; i < n_streams_phimag; i++) {
       hipStreamDestroy(stream[i]);
     }
 
     pb_SwitchToTimer(&timers, pb_TimerID_COPY);
     for (i = 0; i < numK; i++)
       phiMag[i] = phiMag_p[i];
 
     hipHostFree(phiR_p);
     hipHostFree(phiI_p);
     hipHostFree(phiMag_p);
     pb_SwitchToTimer(&timers, pb_TimerID_COPY);
 
   /* Freeing up no longer needed memory on GPU */
   hipFree(phiMag_d);
   hipFree(phiI_d);
   hipFree(phiR_d);
   hipDeviceSynchronize();
 
   pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
 
   kVals = (struct kValues*)calloc(numK, sizeof (struct kValues));
   int k;
   for (k = 0; k < numK; k++) {
     kVals[k].Kx = kx[k];
     kVals[k].Ky = ky[k];
     kVals[k].Kz = kz[k];
     kVals[k].PhiMag = phiMag[k];
   }
  
   pb_SwitchToTimer(&timers, pb_TimerID_COPY);
 
   /* Allocating memory on GPU */
   hipMalloc((void** )&Qr_d, sizeof(float) * numX);
   hipMalloc((void** )&Qi_d, sizeof(float) * numX);
   hipMalloc((void** )&x_d, sizeof(float) * numX);
   hipMalloc((void** )&y_d, sizeof(float) * numX);
   hipMalloc((void** )&z_d, sizeof(float) * numX);
   hipDeviceSynchronize();
 
   {
     /* Allocate pinned memory */
     float *x_p, *y_p, *z_p, *Qr_p, *Qi_p;
 
     hipHostAlloc((void **)&x_p, sizeof(float) * numX, hipHostMallocDefault);
     hipHostAlloc((void **)&y_p, sizeof(float) * numX, hipHostMallocDefault);
     hipHostAlloc((void **)&z_p, sizeof(float) * numX, hipHostMallocDefault);
     hipHostAlloc((void **)&Qr_p, sizeof(float) * numX, hipHostMallocDefault);
     hipHostAlloc((void **)&Qi_p, sizeof(float) * numX, hipHostMallocDefault);
     for (int i = 0; i < numX; i++) {
       x_p[i] = x[i];
       y_p[i] = y[i];
       z_p[i] = z[i];
       Qr_p[i] = Qr[i];
       Qi_p[i] = Qi[i];
     }
 
     int n = 0, i = 0;
     unsigned int offset = 0;
     const unsigned int n_streams_q = 4;
     const unsigned int stream_size = ceil(numX / n_streams_q);
     hipStream_t stream[n_streams_q];
 
     /* Create CUDA streams */
     for (i = 0; i < n_streams_q; i++) {
       hipStreamCreate(&stream[i]);
     }
 
     pb_SwitchToTimer(&timers, pb_TimerID_COPY_ASYNC);
     /* Copy data to the device asynchronously */
     for (n = 0; n < n_streams_q; n++) {
       offset = n * stream_size;
 
       hipMemcpy(&x_d[offset], &x_p[offset],
         sizeof(float) * stream_size, hipMemcpyHostToDevice);
       hipMemcpy(&y_d[offset], &y_p[offset],
         sizeof(float) * stream_size, hipMemcpyHostToDevice);
       hipMemcpy(&z_d[offset], &z_p[offset],
         sizeof(float) * stream_size, hipMemcpyHostToDevice);
       hipMemsetAsync(&Qr_d[offset], 0,
         sizeof(float) * stream_size, stream[n]);
       hipMemsetAsync(&Qi_d[offset], 0,
         sizeof(float) * stream_size, stream[n]);
     }
 
     pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);
     /* Call the kernels with respective streams */
     for (n = 0; n < n_streams_q; n++) {
       offset = n * stream_size;
       ComputeQGPUAsync(numK, stream_size, kVals, x_d, y_d, z_d, Qr_d, Qi_d,
         stream[n], offset);
     }
 
     pb_SwitchToTimer(&timers, pb_TimerID_COPY_ASYNC);
     /* Copy data from the device asynchronously */
     for (n = 0; n < n_streams_q; n++) {
       offset = n * stream_size;
 
       hipMemcpy(&Qr_p[offset], &Qr_d[offset],
         sizeof(float) * stream_size, hipMemcpyDeviceToHost);
       hipMemcpy(&Qi_p[offset], &Qi_d[offset],
         sizeof(float) * stream_size, hipMemcpyDeviceToHost);
     }
     hipDeviceSynchronize();
 
     /* Delete the streams */
     for (i = 0; i < n_streams_q; i++) {
       hipStreamDestroy(stream[i]);
     }
 
     pb_SwitchToTimer(&timers, pb_TimerID_COPY);
     for (i = 0; i < numX; i++) {
       Qr[i] = Qr_p[i];
       Qi[i] = Qi_p[i];
     }
     hipHostFree(x_p);
     hipHostFree(y_p);
     hipHostFree(z_p);
     hipHostFree(Qr_p);
     hipHostFree(Qi_p);
   } 
 
   /* Freeing up no longer needed memory on GPU */
   hipFree(z_d);
   hipFree(y_d);
   hipFree(x_d);
   hipFree(Qi_d);
   hipFree(Qr_d);
   hipDeviceSynchronize();
   hipDeviceReset();
 
   if (params->outFile)
   {
     /* Write Q to file */
     pb_SwitchToTimer(&timers, pb_TimerID_IO);
     outputData(params->outFile, Qr, Qi, numX);
     pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
   }
 
 
   free (kx);
   free (ky);
   free (kz);
   free (x);
   free (y);
   free (z);
   free (phiR);
   free (phiI);
   free (phiMag);
   free (kVals);
   free (Qr);
   free (Qi);
 
   pb_SwitchToTimer(&timers, pb_TimerID_NONE);
   pb_PrintTimerSet(&timers);
   pb_FreeParameters(params);
 
   return 0;
 }
 