#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#define PI   3.1415926535897932384626433832795029f
#define PIx2 6.2831853071795864769252867665590058f

#define MIN(X,Y) ((X) < (Y) ? (X) : (Y))
#define K_ELEMS_PER_GRID 2048

#define PROJECT_DEF 1

struct kValues {
  float Kx;
  float Ky;
  float Kz;
  float PhiMag;
};

#define BLOCK_SIZE 512
#define K_VAL_GRID_SIZE (BLOCK_SIZE * 4)
__constant__ __device__ kValues const_kValues[K_VAL_GRID_SIZE];

__global__ void ComputePhiMagKernelAsync(int numK, float *phiR, float *phiI,
                                         float *phiMag, int offset)
{
  unsigned int t = offset + threadIdx.x + (blockIdx.x * blockDim.x);
  if (t < numK) {
    phiMag[t] = (phiR[t] * phiR[t]) + (phiI[t] * phiI[t]);
  }
}

__global__ void ComputeQKernelAsync(int numK, int numX,
                                    float *x_d, float *y_d, float *z_d,
                                    float *Qr_d, float *Qi_d, int offset)
{
  unsigned int t = threadIdx.x + (blockIdx.x * blockDim.x);

  if (t >= numX)
    return;

  float x_l = x_d[t + offset];
  float y_l = y_d[t + offset];
  float z_l = z_d[t + offset];
  float Qracc = 0.0f;
  float Qiacc = 0.0f;
  float phi = 0.0f;

  float expArg;
  int idx = 0;

  for (; idx < numK; idx++) {
    /* using thread coarsening technique */
    expArg = PIx2 * (const_kValues[idx].Kx * x_l +
                     const_kValues[idx].Ky * y_l +
                     const_kValues[idx].Kz * z_l);

    phi = const_kValues[idx].PhiMag;

    Qracc += phi * cos(expArg);
    Qiacc += phi * sin(expArg);
  }

  Qr_d[t + offset] += Qracc;
  Qi_d[t + offset] += Qiacc;

}

void ComputePhiMagGPUAsync(int numK, int streamSz, float* phiR_d, float* phiI_d,
                           float* phiMag_d, hipStream_t stream, int offset)
{
  unsigned int numBlocks = ((streamSz - 1) / BLOCK_SIZE) + 1;
  dim3 dimGrid(numBlocks, 1, 1);
  dim3 dimBlock(BLOCK_SIZE, 1, 1);

  ComputePhiMagKernelAsync<<<dimGrid, dimBlock, 0, stream>>>(numK,
    phiR_d, phiI_d, phiMag_d, offset);
}

void ComputeQGPUAsync(int numK, int streamSz, struct kValues *kVals,
                      float *x_d, float *y_d, float *z_d,
                      float *Qr_d, float *Qi_d,
                      hipStream_t stream, int offset)
{
  unsigned int kV_size_to_cover = K_VAL_GRID_SIZE;
  unsigned int n_iter = ((numK - 1) / K_VAL_GRID_SIZE) + 1;
  struct kValues *kV_ptr = kVals;

  unsigned int numBlocks = ((streamSz - 1) / BLOCK_SIZE) + 1;
  dim3 dimGrid(numBlocks, 1, 1);
  dim3 dimBlock(BLOCK_SIZE, 1, 1);


  for (int iter = 0; iter < n_iter; iter++) {
    kV_size_to_cover = MIN(K_VAL_GRID_SIZE, numK - (iter * K_VAL_GRID_SIZE));
    if (kV_size_to_cover) {
      hipMemcpyToSymbol(HIP_SYMBOL(const_kValues), kV_ptr,
        kV_size_to_cover * sizeof(struct kValues), 0);

      ComputeQKernelAsync<<<dimGrid, dimBlock, 0, stream>>>(kV_size_to_cover,
        streamSz, x_d, y_d, z_d, Qr_d, Qi_d, offset);
      hipDeviceSynchronize();
    }
    kV_ptr += kV_size_to_cover;
  }

}

void createDataStructsCPU(int numK, int numX, float** phiMag,
	 float** Qr, float** Qi)
{
  *phiMag = (float* ) memalign(16, numK * sizeof(float));
  *Qr = (float*) memalign(16, numX * sizeof (float));
  memset((void *)*Qr, 0, numX * sizeof(float));
  *Qi = (float*) memalign(16, numX * sizeof (float));
  memset((void *)*Qi, 0, numX * sizeof(float));
}
